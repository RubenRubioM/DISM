#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>



void suma_vectores(float *a, float *b, float *c, int n) {

	for (int i = 0; i < n; ++i) {
		c[i] = a[i] + b[i];
	}
}

//Ejecutado en la GPU y llamado por el procesador
__global__
void suma_vectores_gpu(float *a, float *b, float *c, int n) {
	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;
	c[idx_] = a[idx_] + b[idx_];
}


int main(void) {
	hipSetDevice(0);
	const int kNumElements = 25600;
	const int kNumBytes = sizeof(float)*kNumElements;

	float *h_a_ = (float *)malloc(kNumBytes);
	float *h_b_ = (float *)malloc(kNumBytes);
	float *h_c_ = (float *)malloc(kNumBytes);

	if (h_a_ == NULL || h_b_ == NULL || h_c_ == NULL) {
		std::cerr << "Fallo al reservar memoria en host\n";
		getchar();
		exit(-1);
	}

	for (int i = 0; i < kNumElements; ++i) {
		h_a_[i] = rand() / (float)RAND_MAX;
		h_b_[i] = rand() / (float)RAND_MAX;

	}

	//Reservamos memorias en la tarjeta grafica
	float *d_a_ = NULL;
	float *d_b_ = NULL;
	float *d_c_ = NULL;

	hipMalloc((void **)&d_a_, kNumBytes);
	hipMalloc((void **)&d_b_, kNumBytes);
	hipMalloc((void **)&d_c_, kNumBytes);

	//Transferimos a la memoria de la gpu
	hipMemcpy(d_a_, h_a_, kNumBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b_, h_b_, kNumBytes, hipMemcpyHostToDevice);

	int threads_per_block_ = 256;
	int block_per_grid_ = kNumElements / threads_per_block_;

	dim3 tpb_(threads_per_block_, 1, 1);
	dim3 bpg_(block_per_grid_, 1, 1);

	suma_vectores_gpu << <bpg_, tpb_ >> > (d_a_, d_b_, d_c_, kNumElements);
	hipError_t err_ = hipGetLastError();

	if (err_ != hipSuccess) {
		std::cerr << "Fallo al lanzar el kernel con erro" << hipGetErrorString(err_) << "\n";
		getchar();
		exit(-1);
	}

	//copiando de gpu a cpu
	hipMemcpy(h_c_, d_c_, kNumBytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < kNumElements; ++i) {

		if (fabs(h_a_[i] + h_b_[i] - h_c_[i]) > 1e-5) {
			std::cerr << "Error en la posicion " << i << "\n";
			getchar();
			exit(-1);
		}
	}

	hipFree(d_a_);
	hipFree(d_b_);
	hipFree(d_c_);

	free(h_a_);
	free(h_b_);
	free(h_c_);

	hipDeviceReset();

	std::cout << "Exito\n";
	std::cout << *h_c_;
	getchar();
	return 0;


}