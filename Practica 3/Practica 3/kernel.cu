
#include "hip/hip_runtime.h"


#include <iostream>


//Como se haria de manera tradicional
void suma_vectores(float *a, float *b, float *c, int n) {

	for (int i = 0; i < n; ++i) {
		c[i] = a[i] + b[i];
	}
}

//Ejecutado en la GPU y llamado por el procesador
__global__
void suma_vectores_gpu(float *a, float *b, float *c, int n) {
	// threadIdx.x --> nos dice dentro de cada bloque que hilo soy
	// blockIdx.x --> nos dice que bloque soy dentro de una malla
	// blockDim.x --> tamanyo del bloque

	//Ejercicio 3
	
	int idx_ = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx_ < n) {
		
		c[idx_] = a[idx_] + b[idx_];
		
	}
	

	//Ejercicio 4
	
	/*int idx_ = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx_ < n) {
		for (int i = idx_; i < n; i += blockDim.x * gridDim.x)
		{
			c[i] = a[i] + b[i];
		}
	}*/
	
	
}


int main(void) {
	
	hipSetDevice(0);
	const int kNumElements = 30000000;
	const int kNumBytes = sizeof(float)*kNumElements;
	

	float *h_a_ = (float *)malloc(kNumBytes);
	float *h_b_ = (float *)malloc(kNumBytes);
	float *h_c_ = (float *)malloc(kNumBytes);

	if (h_a_ == NULL || h_b_ == NULL || h_c_ == NULL) {
		std::cerr << "Fallo al reservar memoria en host\n";
		getchar();
		exit(-1);
	}
	
	for (int i = 0; i < kNumElements; ++i) {
		h_a_[i] = rand() / (float)RAND_MAX;
		h_b_[i] = rand() / (float)RAND_MAX;

	}

	//Reservamos memorias en la tarjeta grafica
	float *d_a_ = NULL;
	float *d_b_ = NULL;
	float *d_c_ = NULL;

	hipMalloc((void **)&d_a_, kNumBytes);
	hipMalloc((void **)&d_b_, kNumBytes);
	hipMalloc((void **)&d_c_, kNumBytes);

	//Transferimos a la memoria de la gpu
	hipMemcpy(d_a_, h_a_, kNumBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b_, h_b_, kNumBytes, hipMemcpyHostToDevice);

	int hilosPorBloque = 256;
	int bloquesPorGrid;
	
	//int bloquesPorGrid = (int)ceil((kNumElements / hilosPorBloque));
	if (kNumElements%hilosPorBloque == 0) {
		bloquesPorGrid = (kNumElements / hilosPorBloque);
	}
	else {
		bloquesPorGrid = (kNumElements / hilosPorBloque)+1;
	}

	
	//Sirve para calcular el numero maximo de bloques que se puede crear en un grid simultaneamente
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	
	dim3 tpb_(hilosPorBloque, 1, 1);
	dim3 bpg_(bloquesPorGrid, 1, 1);
	
	suma_vectores_gpu <<<bpg_, tpb_ >>> (d_a_, d_b_, d_c_, kNumElements);

	hipError_t err_ = hipGetLastError();

	if (err_ != hipSuccess) {
		std::cerr << "Fallo al lanzar el kernel con erro" << hipGetErrorString(err_) << "\n";
		getchar();
		exit(-1);
	}

	//copiando de gpu a cpu
	hipMemcpy(h_c_, d_c_, kNumBytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < kNumElements; ++i) {

		if (fabs(h_a_[i] + h_b_[i] - h_c_[i]) > 1e-5) {
			std::cerr << "Error en la posicion " << i << "\n";
			getchar();
			exit(-1);
		}
	}

	hipFree(d_a_);
	hipFree(d_b_);
	hipFree(d_c_);
	
	free(h_a_);
	free(h_b_);
	free(h_c_);

	hipDeviceReset();


	std::cout << "Exito\n";
	
	getchar();
	return 0;


}